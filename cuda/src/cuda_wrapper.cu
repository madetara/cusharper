#include "cuda_wrapper.cuh"

DLLEXPORT void check_cusolver_sp() {
    hipsolverSpHandle_t handle;
    CUSOLVER_CHECK(hipsolverSpCreate(&handle));
    CUSOLVER_CHECK(hipsolverSpDestroy(handle));
}
